#include "hip/hip_runtime.h"
#include <fmt/base.h>

#include "codegen/module.hpp"
#include "runtime/Runtime.hpp"
#include "runtime/UserProvided.hpp"
#include "runtime/Utils.hpp"

__global__ void cudalator_apply_input(StateType *dut, int cycle, size_t len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= len) return;

    cu::Random rng(cycle ^ tid);

    if (cycle % 2 == 0) {
        dut[tid].CK = 0;
        return;
    }

    #define DRIVE(input) dut[tid].input = (rng.get() % 2)
    dut[tid].CK = 1;

    DRIVE(n3065gat);
    DRIVE(n3066gat);
    DRIVE(n3067gat);
    DRIVE(n3068gat);
    DRIVE(n3069gat);
    DRIVE(n3070gat);
    DRIVE(n3071gat);
    DRIVE(n3072gat);
    DRIVE(n3073gat);
    DRIVE(n3074gat);
    DRIVE(n3075gat);
    DRIVE(n3076gat);
    DRIVE(n3077gat);
    DRIVE(n3078gat);
    DRIVE(n3079gat);
    DRIVE(n3080gat);
    DRIVE(n3081gat);
    DRIVE(n3082gat);
    DRIVE(n3083gat);
    DRIVE(n3084gat);
    DRIVE(n3085gat);
    DRIVE(n3086gat);
    DRIVE(n3087gat);
    DRIVE(n3088gat);
    DRIVE(n3089gat);
    DRIVE(n3090gat);
    DRIVE(n3091gat);
    DRIVE(n3092gat);
    DRIVE(n3093gat);
    DRIVE(n3094gat);
    DRIVE(n3095gat);
    DRIVE(n3097gat);
    DRIVE(n3098gat);
    DRIVE(n3099gat);
    DRIVE(n3100gat);
}

int main(int argc, char *argv[]) {
    size_t circuit_number = 256;
    if (argc == 2) {
        circuit_number = atoi(argv[1]);
    }

    Circuit circuit(circuit_number);

    constexpr int MAX = 1000;

    // circuit.open_vcd("waves.vcd", 3);
    fmt::println("Starting Simulation");
    for (int i = 0; i < MAX; i++) {
        cu::progress_display(i, MAX);

        circuit.apply_input();
        circuit.eval();
    }
    fmt::println("\nSimulation Done!");

    circuit.get_stats().print();
    circuit.get_stats().save_to_json("result.json");
}

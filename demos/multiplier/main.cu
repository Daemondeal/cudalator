#include "hip/hip_runtime.h"
#include <fmt/base.h>

#include "codegen/module.hpp"
#include "runtime/Runtime.hpp"
#include "runtime/UserProvided.hpp"

struct xorshift32_state {
    uint32_t a;
};

/* The state must be initialized to non-zero */
__device__ uint32_t xorshift32(xorshift32_state *state)
{
    /* Algorithm "xor" from p. 4 of Marsaglia, "Xorshift RNGs" */
    uint32_t x = state->a;
    x ^= x << 13;
    x ^= x >> 17;
    x ^= x << 5;
    return state->a = x;
}


__global__ void cudalator_apply_input(StateType *dut, int cycle, size_t len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= len) return;

    int seed = cycle ^ tid;
    xorshift32_state rng;
    rng.a = seed;
    xorshift32(&rng);
    uint32_t a = xorshift32(&rng);
    uint32_t b = xorshift32(&rng);

    dut[tid].i_A = Bit<8>(a % 256);
    dut[tid].i_B = Bit<8>(b % 256);
}

int main(int argc, char *argv[]) {
    size_t circuit_number = 256;
    if (argc == 2) {
        circuit_number = atoi(argv[1]);
    }

    Circuit circuit(circuit_number);

    constexpr int MAX = 1000;
    constexpr int STEP = std::max(MAX / 1000, 1);

    // circuit.open_vcd("waves.vcd", 3);
    fmt::println("Starting Simulation");
    for (int i = 0; i < MAX; i++) {
        if (i % STEP == 0) {
            int steps = (20 * (i+1)) / MAX;

            fmt::print("\r[");
            for (int j = 0; j < steps; j++) {
                fmt::print("#");
            }

            for (int j = steps; j < 20; j++) {
                fmt::print(".");
            }
            fmt::print("] {}/{}", i+1, MAX);
            fflush(stdout);
        }


        circuit.apply_input();
        circuit.eval();
    }
    fmt::println("");
    fmt::println("Simulation Done!");

    circuit.get_stats().print();
    circuit.get_stats().save_to_json("result.json");
}

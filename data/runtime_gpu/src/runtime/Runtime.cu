#include "Runtime.hpp"
#include "Stats.hpp"
#include "cuda_compat.hpp"
#include "UserProvided.hpp"
#include <vector>

Circuit::Circuit(int number_of_circuits)
    : m_num_circuits(number_of_circuits), m_cycles(0), m_stats({}) {
    // allocate on the gpu
    size_t states_size = sizeof(StateType) * m_num_circuits;
    size_t diffs_size = sizeof(DiffType) * m_num_circuits;
    CUDA_CHECK(hipMalloc(&d_states, states_size));
    CUDA_CHECK(hipMalloc(&d_previous_states, states_size));
    CUDA_CHECK(hipMalloc(&d_diffs, diffs_size));

    // apparently a locked page allows for faster transfers
    CUDA_CHECK(hipHostAlloc(&h_diffs, diffs_size, hipHostMallocDefault));
    CUDA_CHECK(hipMemset(d_states, 0, states_size));
    CUDA_CHECK(hipMemset(d_previous_states, 0, states_size));

    m_stats.number_of_circuits = m_num_circuits;
    m_stats.state_array_size = states_size * 2;
    m_stats.diff_array_size = diffs_size;

    m_processes = make_processes();
    first_eval();
}

Circuit::~Circuit() {
    hipFree(d_states);
    hipFree(d_previous_states);
    hipFree(d_diffs);
    hipHostFree(h_diffs);
}

// We copy data to the GPU, we modify it and then copy it back
void Circuit::apply_input() {
    const int threads_per_block = 256;
    const int blocks =
        (m_num_circuits + threads_per_block - 1) / threads_per_block;

    cudalator_apply_input<<<blocks, threads_per_block>>>(d_states, m_cycles, m_num_circuits);
}

void Circuit::eval() {
    std::vector<ProcType> ready_queue;
    const int threads_per_block = 256;
    const int blocks =
        (m_num_circuits + threads_per_block - 1) / threads_per_block;

    m_stats.start_counter(PerfEvent::DoDeltaCycle);
    while (true) {
        m_stats.start_counter(PerfEvent::DoIteration);

        m_stats.start_counter(PerfEvent::CalculateStateDiff);
        // launching the diff kernel
        state_calculate_diff<<<blocks, threads_per_block>>>(
            d_previous_states, d_states, d_diffs, m_num_circuits);
        CUDA_CHECK(hipGetLastError());
        m_stats.stop_counter(PerfEvent::CalculateStateDiff);

        m_stats.start_counter(PerfEvent::PopulateReadyQueue);
        // coying the diff results
        CUDA_CHECK(hipMemcpy(h_diffs, d_diffs,
                              sizeof(DiffType) * m_num_circuits,
                              hipMemcpyDeviceToHost));
        m_stats.stop_counter(PerfEvent::PopulateReadyQueue);

        m_stats.start_counter(PerfEvent::PopulateReadyQueue);
        // cpu computation of the ready queue
        for (const auto& proc : m_processes) {
            bool should_run = false;
            for (int c = 0; c < m_num_circuits && !should_run; ++c) {
                const DiffType& diff = h_diffs[c];
                for (auto [signal_idx, change_type] : proc.sensitivity) {
                    auto actual = diff.change[signal_idx];
                    if ((change_type == ChangeType::Change &&
                         actual != ChangeType::NoChange) ||
                        (change_type == actual)) {
                        should_run = true;
                        break;
                    }
                }
            }
            if (should_run)
                ready_queue.push_back(proc);
        }
        m_stats.stop_counter(PerfEvent::PopulateReadyQueue);

        // if empy, it's already stable
        if (ready_queue.empty()) {
            m_stats.stop_counter(PerfEvent::DoIteration);
            break;
        }
        m_stats.kernels_launched += ready_queue.size();

        m_stats.start_counter(PerfEvent::CloneStates);
        // save current state for next diff computation
        CUDA_CHECK(hipMemcpy(d_previous_states, d_states,
                              sizeof(StateType) * m_num_circuits,
                              hipMemcpyDeviceToDevice));
        m_stats.stop_counter(PerfEvent::CloneStates);

        m_stats.start_counter(PerfEvent::RunKernels);
        // launch of all the ready kernels
        for (const auto& proc : ready_queue) {
            run_process<<<blocks, threads_per_block>>>(d_states, m_num_circuits,
                                                       proc.id);
            CUDA_CHECK(hipGetLastError());
        }
        m_stats.stop_counter(PerfEvent::RunKernels);

        ready_queue.clear();
        m_stats.stop_counter(PerfEvent::DoIteration);
        m_stats.iterations_done++;
    }
    // end of delta cycle step

    m_cycles++;
    m_stats.delta_times_ran++;

    // state copy before next eval call
    CUDA_CHECK(hipMemcpy(d_previous_states, d_states,
                          sizeof(StateType) * m_num_circuits,
                          hipMemcpyDeviceToDevice));
    m_stats.stop_counter(PerfEvent::DoDeltaCycle);

    dump_to_vcd();
}

void Circuit::first_eval() {
    const int threads_per_block = 256;
    const int blocks =
        (m_num_circuits + threads_per_block - 1) / threads_per_block;
    for (const auto& proc : m_processes) {
        run_process<<<blocks, threads_per_block>>>(d_states, m_num_circuits,
                                                   proc.id);
    }
    CUDA_CHECK(hipDeviceSynchronize());
    eval();
    m_cycles--;
    m_stats.kernels_launched += m_processes.size();
}

// no need to do anything below here

void Circuit::open_vcd(const std::string& path, int circuit_idx) {
    auto fp = std::make_unique<fmt::ostream>(fmt::output_file(path));
    m_vcd.emplace(std::move(fp), circuit_idx);
    m_vcd->print_header();
}

void Circuit::dump_to_vcd() {
    if (m_vcd.has_value()) {
        std::vector<StateType> h_states(m_num_circuits);
        CUDA_CHECK(hipMemcpy(h_states.data(), d_states,
                              sizeof(StateType) * m_num_circuits,
                              hipMemcpyDeviceToHost));
        m_vcd->dump(h_states, m_cycles);
    }
}
